#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>

#include "include/defines.h"
#include "kernels/kernels.h"
#include "futils/futils.h"

int main() {
    std::cout << N_PARTICLES << " particles." << std::endl;

    // calculate the number of blocks needed
    constexpr int blocks_per_row = N_PARTICLES % BLOCK_SIZE == 0 ?
                                       N_PARTICLES / BLOCK_SIZE : N_PARTICLES / BLOCK_SIZE + 1;
    constexpr int n_blocks = blocks_per_row * (blocks_per_row + 1) / 2;

    // allocate pos, vel
    auto *pos = static_cast<double *>(malloc(N_PARTICLES * DIM * sizeof(double)));
    auto *vel = static_cast<double *>(malloc(N_PARTICLES * DIM * sizeof(double)));

    // allocate matrix, pos, and vel array on device
    double *device_matrix, *device_pos, *device_vel;
    hipMalloc(&device_matrix, N_PARTICLES*N_PARTICLES*sizeof(double)); checkCudaError("cudaMalloc1");
    hipMalloc(&device_pos, N_PARTICLES * DIM * sizeof(double));        checkCudaError("cudaMalloc2");
    hipMalloc(&device_vel, N_PARTICLES * DIM * sizeof(double));        checkCudaError("cudaMalloc2.5");

    // fill initial conditions
    fill_array(pos, N_PARTICLES*DIM);
    fill_array(vel, N_PARTICLES*DIM);

    // copy initial condition on deviuce
    hipMemcpy(device_pos, pos, N_PARTICLES * DIM * sizeof(double), hipMemcpyHostToDevice); checkCudaError("cudaMalloc4");
    hipMemcpy(device_vel, vel, N_PARTICLES * DIM * sizeof(double), hipMemcpyHostToDevice); checkCudaError("cudaMalloc4.5");

    // allocate force array on device
    double *device_force;
    hipMalloc(&device_force, N_PARTICLES * DIM * sizeof(double));

    // launch kernel
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_dim(n_blocks);

    for (int cmp = 0; cmp < DIM; cmp++) {
        calculate_pairwise_force_component<<<grid_dim, block_dim>>>(device_pos, cmp, device_matrix, N_PARTICLES, blocks_per_row);
        checkCudaError("kernel 1 launch");
        hipDeviceSynchronize();

        // calculate total force (1 component) on each particle
        sum_over_rows<<<n_blocks, BLOCK_SIZE>>>(device_matrix, device_force + (cmp * N_PARTICLES), N_PARTICLES);
        checkCudaError("kernel 2 launch");
        hipDeviceSynchronize();
    }

    // apply force on particles
    apply_motion<<<N_PARTICLES % 1024 == 0 ? N_PARTICLES / 1024 : N_PARTICLES / 1024 + 1,1024>>>(device_pos, device_vel, device_force, N_PARTICLES, forwardEuler, D_T);
    hipDeviceSynchronize();

    // copy back new position
    hipMemcpy(pos, device_pos, N_PARTICLES * DIM * sizeof(double), hipMemcpyDeviceToHost); checkCudaError("cudaMalloc9");
    hipMemcpy(vel, device_vel, N_PARTICLES * DIM * sizeof(double), hipMemcpyDeviceToHost); checkCudaError("cudaMalloc9.5");


    // free space on Host and device
    hipFree(device_matrix);

    free(pos);  hipFree(device_pos);
    free(vel);  hipFree(device_vel);

    hipFree(device_force);

    return EXIT_SUCCESS;
}